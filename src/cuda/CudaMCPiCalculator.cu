#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "mc/MCPiCalculator.h"
#include "random/Random.hpp"

__global__ void MCKernel(unsigned long long* d_inside, unsigned long long chunkSamples,
                         unsigned int seed) {
    unsigned long long localCount = 0;
    hiprandState        state;
    hiprand_init(seed, threadIdx.x + blockIdx.x * blockDim.x, 0, &state);
    for (unsigned long long i = 0; i < chunkSamples; ++i) {
        float x = hiprand_uniform(&state) * 2.0f - 1.0f;
        float y = hiprand_uniform(&state) * 2.0f - 1.0f;
        if (x * x + y * y <= 1.0f)
            localCount++;
    }
    atomicAdd(d_inside, localCount);
}

class CUDAMCPiCalculator : public IMCPiCalculator {
   public:
    long double estimatePi(unsigned long long totalSamples, int threadCount,
                           unsigned long long chunkSize, RNGType rngType,
                           DistType distType) override {
        unsigned long long  h_inside = 0;
        unsigned long long* d_inside;
        hipMalloc(&d_inside, sizeof(unsigned long long));
        hipMemcpy(d_inside, &h_inside, sizeof(unsigned long long), hipMemcpyHostToDevice);

        int threadsPerBlock = 1024;
        int blocks          = (threadCount + threadsPerBlock - 1) / threadsPerBlock;

        std::random_device rd;
        unsigned int       seed = rd() + 42;
        MCKernel<<<blocks, threadsPerBlock>>>(d_inside, chunkSize, seed);
        hipDeviceSynchronize();

        hipMemcpy(&h_inside, d_inside, sizeof(unsigned long long), hipMemcpyDeviceToHost);
        hipFree(d_inside);
        return (long double)4.0 * h_inside / totalSamples;
    }
};

IMCPiCalculator* createCUDACalculator() {
    return new CUDAMCPiCalculator();
}
