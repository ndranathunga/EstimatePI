#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "mc/MCPiCalculator.h"
#include "random/Random.hpp"

__global__ void MCKernel(unsigned long long* d_inside, long long chunkSamples, unsigned int seed) {
    unsigned long long localCount = 0;
    hiprandState        state;
    hiprand_init(seed, threadIdx.x + blockIdx.x * blockDim.x, 0, &state);
    for (long long i = 0; i < chunkSamples; ++i) {
        float x = hiprand_uniform(&state) * 2.0f - 1.0f;
        float y = hiprand_uniform(&state) * 2.0f - 1.0f;
        if (x * x + y * y <= 1.0f)
            localCount++;
    }
    atomicAdd(d_inside, localCount);
}

class CUDAMCPiCalculator : public IMCPiCalculator {
   public:
    double estimatePi(long long totalSamples, int threadCount, long long chunkSize, RNGType rngType,
                      DistType distType) override {
        // FIXME: ChatGPT generated temporary code
        unsigned long long  h_inside = 0;
        unsigned long long* d_inside;
        hipMalloc(&d_inside, sizeof(unsigned long long));
        hipMemcpy(d_inside, &h_inside, sizeof(unsigned long long), hipMemcpyHostToDevice);

        int threadsPerBlock = 256;
        int blocks          = (threadCount + threadsPerBlock - 1) / threadsPerBlock;
        MCKernel<<<blocks, threadsPerBlock>>>(d_inside, chunkSize, 42);
        hipDeviceSynchronize();

        hipMemcpy(&h_inside, d_inside, sizeof(unsigned long long), hipMemcpyDeviceToHost);
        hipFree(d_inside);
        return 4.0 * h_inside / totalSamples;
    }
};

IMCPiCalculator* createCUDACalculator() {
    return new CUDAMCPiCalculator();
}
